#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void print_threadIdx() {
  printf("threadIdx.x: %d, threadIdx.y: %d, threadIdx.z: %d\n", threadIdx.x,
         threadIdx.y, threadIdx.z);
}

__global__ void print_blockIdx_and_gridIdx() {
  printf(
      "blockIdx.x: %d, blockIdx.y: %d, blockIdx.z: %d, gridDim.x: %d, "
      "gridDim.y: %d, gridDim.z: %d\n",
      blockIdx.x, blockIdx.y, blockIdx.z, gridDim.x, gridDim.y, gridDim.z);
}

//int main() {
//  int nx, ny, nz;
//  nx = 4;
//  ny = 4;
//  nz = 4;
//  dim3 block(2, 2, 2);
//  dim3 grid(nx / block.x, ny / block.y);
// /* print_threadIdx<<<grid, block>>>();*/
//  print_blockIdx_and_gridIdx<<<grid, block>>>();
//  cudaDeviceSynchronize();
//  cudaDeviceReset();
//  return 0;
//}